#include "hip/hip_runtime.h"
/*
   Copyright 2015 Kyuyeon Hwang (kyuyeon.hwang@gmail.com)

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/


#include "CudaKernels.h"

#define THREAD_PER_BLOCK 512

namespace fractal
{

namespace cudaKernels
{


template<class T>
inline __device__ T _exp(const T x);

template<class T>
inline __device__ T _log(const T x);

template<class T>
inline __device__ T _sqrt(const T x);

template<class T>
static __global__ void MatSetKernel(T *x, const int ldx, const T val,
        const int nRows, const int nCols);

template<class T>
static __global__ void MatElemMultKernel(const T *x, const int ldx,
        const T *y, const int ldy, T *z, const int ldz,
        const int nRows, const int nCols);

template<class T>
static __global__ void MatAddKernel(const T *x, const int ldx,
        const T *y, const int ldy, T *z, const int ldz,
        const int nRows, const int nCols);

template<class T>
static __global__ void MatSubKernel(const T *x, const int ldx,
        const T *y, const int ldy, T *z, const int ldz,
        const int nRows, const int nCols);

template<class T>
static __global__ void MatAddToDiagKernel(T *x, const T val,
        const int shift, const int nRows);

template<class T>
static __global__ void MatMakeTriKernel(T *x, const bool upper, const int nRows);

template<class T>
static __global__ void FuncSigmoidKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols);

template<class T>
static __global__ void FuncTanhKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols);

template<class T>
static __global__ void FuncSoftplusKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols);

template<class T>
static __global__ void FuncRectLinearKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols);

template<class T>
static __global__ void FuncSoftmaxKernel(const T *x, const int ldx,
        T *y, const int ldy, const int n);

template<class I, class V>
static __global__ void FuncCTCDecodeKernel(const V *x, const int ldx,
        V *y, const int ldy, const I *prevIdxMax, I *idxMax,
        const int layerSize, const int nStep);

template<class T>
static __global__ void FuncBoundRangeKernel(const T *x, const int ldx,
        T *y, const int ldy, const T _min, const T _max,
        const int nRows, const int nCols);

template<class T>
static __global__ void FuncSigmoidDerivKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols);

template<class T>
static __global__ void FuncTanhDerivKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols);

template<class T>
static __global__ void FuncSoftplusDerivKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols);

template<class T>
static __global__ void FuncRectLinearDerivKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols);

template<class I, class V>
static __global__ void OneHotEncodeKernel(const I *index, V *vector, const int n);

template<class T>
static __global__ void GenerateDropoutMaskKernel(T *mask, const T *uniformDist,
        const int n, const T dropoutRate);

template<class T>
static __global__ void RmspropKernel(T *newDerivs, const T *derivs, T *msDeriv,
        const T decayRate, const int n);

template<class T>
static __global__ void AdadeltaKernel(T *deltas, const T *derivs, T *msDeriv, T *msDelta,
        const T learningRate, const T decayRate, const int n);


template<>
inline __device__ float _exp<float>(const float x)
{
    return min(__expf(x), 1e30);
}


template<>
inline __device__ double _exp<double>(const double x)
{
    return min(exp(x), 1e300);
}


template<>
inline __device__ float _log<float>(const float x)
{
    return __logf(x);
}


template<>
inline __device__ double _log<double>(const double x)
{
    return log(x);
}


template<>
inline __device__ float _sqrt<float>(const float x)
{
    return __fsqrt_rn(x);
}


template<>
inline __device__ double _sqrt<double>(const double x)
{
    return __dsqrt_rn(x);
}



template<class T>
static __global__ void MatSetKernel(T *x, const int ldx,
        const T val, const int nRows, const int nCols)
{
    int thdIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int col = thdIdx / nRows;
    int row = thdIdx - col * nRows;
    int xIdx = col * ldx + row;

    if(thdIdx >= nRows * nCols) return;

    x[xIdx] = val;
}


template<class T>
static __global__ void MatElemMultKernel(const T *x, const int ldx,
        const T *y, const int ldy, T *z, const int ldz, const int nRows, const int nCols)
{
    int thdIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int col = thdIdx / nRows;
    int row = thdIdx - col * nRows;
    int xIdx = col * ldx + row;
    int yIdx = col * ldy + row;
    int zIdx = col * ldz + row;

    if(thdIdx >= nRows * nCols) return;

    z[zIdx] = x[xIdx] * y[yIdx];
}


template<class T>
static __global__ void MatAddKernel(const T *x, const int ldx,
        const T *y, const int ldy, T *z, const int ldz, const int nRows, const int nCols)
{
    int thdIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int col = thdIdx / nRows;
    int row = thdIdx - col * nRows;
    int xIdx = col * ldx + row;
    int yIdx = col * ldy + row;
    int zIdx = col * ldz + row;

    if(thdIdx >= nRows * nCols) return;

    z[zIdx] = x[xIdx] + y[yIdx];
}


template<class T>
static __global__ void MatSubKernel(const T *x, const int ldx,
        const T *y, const int ldy, T *z, const int ldz, const int nRows, const int nCols)
{
    int thdIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int col = thdIdx / nRows;
    int row = thdIdx - col * nRows;
    int xIdx = col * ldx + row;
    int yIdx = col * ldy + row;
    int zIdx = col * ldz + row;

    if(thdIdx >= nRows * nCols) return;

    z[zIdx] = x[xIdx] - y[yIdx];
}


template<class T>
static __global__ void MatAddToDiagKernel(T *x, const T val,
        const int shift, const int nRows)
{
    int idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= nRows) return;

    idx = (idx * (nRows + 1) + shift) % (nRows * nRows);

    x[idx] += val;
}


template<class T>
static __global__ void MatMakeTriKernel(T *x, const bool upper, const int nRows)
{
    int idx;
    int iRow, iCol;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= nRows * nRows) return;

    /* Column-major order */

    iCol = idx / nRows;
    iRow = idx % nRows;

    if(upper == true) /* Make upper triangular matrix */
    {
        x[idx] *= (T) (iCol >= iRow);
    }
    else
    {
        x[idx] *= (T) (iCol <= iRow);
    }
}


template<class T>
static __global__ void FuncSigmoidKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols)
{
    int thdIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int col = thdIdx / nRows;
    int row = thdIdx - col * nRows;
    int xIdx = col * ldx + row;
    int yIdx = col * ldy + row;

    if(thdIdx >= nRows * nCols) return;

    y[yIdx] = (T)1 / ((T)1 + _exp<T>(-x[xIdx]));
}


template<class T>
static __global__ void FuncTanhKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols)
{
    int thdIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int col = thdIdx / nRows;
    int row = thdIdx - col * nRows;
    int xIdx = col * ldx + row;
    int yIdx = col * ldy + row;

    if(thdIdx >= nRows * nCols) return;

    T v = _exp<T>((T)(-2) * x[xIdx]);

    y[yIdx] = (T)2 / ((T)1 + v) - (T)1;
}


template<class T>
static __global__ void FuncSoftplusKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols)
{
    int thdIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int col = thdIdx / nRows;
    int row = thdIdx - col * nRows;
    int xIdx = col * ldx + row;
    int yIdx = col * ldy + row;

    if(thdIdx >= nRows * nCols) return;

    y[yIdx] = _log<T>((T)1 + _exp<T>(x[xIdx]));
}


template<class T>
static __global__ void FuncRectLinearKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols)
{
    int thdIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int col = thdIdx / nRows;
    int row = thdIdx - col * nRows;
    int xIdx = col * ldx + row;
    int yIdx = col * ldy + row;

    if(thdIdx >= nRows * nCols) return;

    //y[yIdx] = max((T)0, x[xIdx]);

    /* Leaky */
    y[yIdx] = max((T)0.01 * x[xIdx], x[xIdx]);
}


template<class T>
static __global__ void FuncSoftmaxKernel(const T *x, const int ldx,
        T *y, const int ldy, const int n)
{
    __shared__ T _v[THREAD_PER_BLOCK];
    T v_tmp, v_max;
    int i;

    x += blockIdx.x * ldx;
    y += blockIdx.x * ldy;


    /* Sequential reduction(max) */
    v_tmp = threadIdx.x < n ? x[threadIdx.x] : (FLOAT) 0;

    #pragma unroll
    for(i = threadIdx.x + blockDim.x; i < n; i += blockDim.x)
    {
        v_tmp = max(v_tmp, x[i]);
    }

    _v[threadIdx.x] = v_tmp;

    __syncthreads();

    /* Parallel reduction(max) */
    i = (blockDim.x >> 1);

    for(; i > 0; i >>= 1)
    {
        if(threadIdx.x < i && threadIdx.x + i < n)
        {
            v_tmp = max(v_tmp, _v[threadIdx.x + i]);
            _v[threadIdx.x] = v_tmp;
        }

        __syncthreads();
    }

    v_max = _v[0];

    __syncthreads();

    /* Sequential reduction(+) */
    v_tmp = (T) 0;

    #pragma unroll
    for(i = threadIdx.x; i < n; i += blockDim.x)
    {
        v_tmp += _exp<T>(x[i] - v_max);
    }

    _v[threadIdx.x] = v_tmp;

    __syncthreads();

    /* Parallel reduction(+) */
    i = (blockDim.x >> 1);
    if(threadIdx.x < i)
        v_tmp = _v[threadIdx.x];

    for(; i > 0; i >>= 1)
    {
        if(threadIdx.x < i)
        {
            v_tmp += _v[threadIdx.x + i];
            _v[threadIdx.x] = v_tmp;
        }

        __syncthreads();
    }


    /* Update */
    v_tmp = _v[0];

    #pragma unroll
    for(i = threadIdx.x; i < n; i += blockDim.x)
    {
        y[i] = _exp<T>(x[i] - v_max) / v_tmp;
    }
}


template<class I, class V>
static __global__ void FuncCTCDecodeKernel(const V *x, const int ldx,
        V *y, const int ldy, const I *prevIdxMax, I *idxMax,
        const int layerSize, const int nStep)
{
    /* One thread block per data stream.
     * Stream index: blockIdx.x
     * Number of streams: gridDim.x
     */

    __shared__ V _v[THREAD_PER_BLOCK];
    __shared__ int _i[THREAD_PER_BLOCK];

    V v_tmp;
    int i_tmp, i_max;
    int i;
    int _prevIdxMax;

    x += blockIdx.x * ldx;
    y += blockIdx.x * ldy;
    idxMax += blockIdx.x;

    if(threadIdx.x == 0)
    {
        _prevIdxMax = prevIdxMax[blockIdx.x];
    }

    __syncthreads();

    for(int curStep = 0; curStep < nStep; curStep++)
    {
        /* Sequential reduction(max) */
        v_tmp = threadIdx.x < layerSize ? x[threadIdx.x] : (FLOAT) 0;
        i_tmp = threadIdx.x < layerSize ? (INT) threadIdx.x : (INT) -1;

        #pragma unroll
        for(i = threadIdx.x + blockDim.x; i < layerSize; i += blockDim.x)
        {
            int comp = (v_tmp > x[i]);

            v_tmp = comp * v_tmp + (1 - comp) * x[i];
            i_tmp = comp * i_tmp + (1 - comp) * i;
        }

        _v[threadIdx.x] = v_tmp;
        _i[threadIdx.x] = i_tmp;

        __syncthreads();

        /* Parallel reduction(max) */
        i = (blockDim.x >> 1);

        for(; i > 0; i >>= 1)
        {
            if(threadIdx.x < i && threadIdx.x + i < layerSize)
            {
                FLOAT v_comp = _v[threadIdx.x + i];
                int i_comp = _i[threadIdx.x + i];

                int comp = (v_tmp > v_comp);

                v_tmp = comp * v_tmp + (1 - comp) * v_comp;
                i_tmp = comp * i_tmp + (1 - comp) * i_comp;

                _v[threadIdx.x] = v_tmp;
                _i[threadIdx.x] = i_tmp;
            }

            __syncthreads();
        }

        //v_max = _v[0];
        i_max = _i[0];

        __syncthreads();


        /* Sequential update */
        #pragma unroll
        for(i = threadIdx.x; i < layerSize - 1; i += blockDim.x)
        {
            y[i] = (i == i_max);
        }

        /* Update clock signal */
        if(threadIdx.x == 0)
        {
            y[layerSize - 1] = (i_max != layerSize - 1) && (i_max != _prevIdxMax);
            *idxMax = i_max;
            _prevIdxMax = i_max;
        }

        __syncthreads();

        /* Next time step */
        x += gridDim.x * ldx;
        y += gridDim.x * ldy;
        idxMax += gridDim.x;
    }
}


template<class T>
static __global__ void FuncBoundRangeKernel(const T *x, const int ldx,
        T *y, const int ldy, const T _min, const T _max,
        const int nRows, const int nCols)
{
    int thdIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int col = thdIdx / nRows;
    int row = thdIdx - col * nRows;
    int xIdx = col * ldx + row;
    int yIdx = col * ldy + row;

    if(thdIdx >= nRows * nCols) return;

    y[yIdx] = min(_max, max(_min, x[xIdx]));
}


template<class T>
static __global__ void FuncSigmoidDerivKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols)
{
    int thdIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int col = thdIdx / nRows;
    int row = thdIdx - col * nRows;
    int xIdx = col * ldx + row;
    int yIdx = col * ldy + row;

    if(thdIdx >= nRows * nCols) return;

    T v = x[xIdx];
    y[yIdx] = v * ((T)1 - v);
}


template<class T>
static __global__ void FuncTanhDerivKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols)
{
    int thdIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int col = thdIdx / nRows;
    int row = thdIdx - col * nRows;
    int xIdx = col * ldx + row;
    int yIdx = col * ldy + row;

    if(thdIdx >= nRows * nCols) return;

    T v = x[xIdx];
    y[yIdx] = ((T)1 - v) * ((T)1 + v);
}


template<class T>
static __global__ void FuncSoftplusDerivKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols)
{
    int thdIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int col = thdIdx / nRows;
    int row = thdIdx - col * nRows;
    int xIdx = col * ldx + row;
    int yIdx = col * ldy + row;

    if(thdIdx >= nRows * nCols) return;

    y[yIdx] = (T)1 - _exp<T>(-x[xIdx]);
}


template<class T>
static __global__ void FuncRectLinearDerivKernel(const T *x, const int ldx,
        T *y, const int ldy, const int nRows, const int nCols)
{
    int thdIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int col = thdIdx / nRows;
    int row = thdIdx - col * nRows;
    int xIdx = col * ldx + row;
    int yIdx = col * ldy + row;

    if(thdIdx >= nRows * nCols) return;

    //y[yIdx] = (T)(x[xIdx] > (T)0);
    /* Leaky */
    y[yIdx] = (T)0.01 + (T)0.99 * (T)(x[xIdx] > (T)0);
}


template<class I, class V>
static __global__ void OneHotEncodeKernel(const I *index, V *vector, const int n)
{
    int elemIdx, batchIdx;
    __shared__ I _index;

    elemIdx = blockIdx.x * blockDim.x + threadIdx.x;
    batchIdx = blockIdx.y;

    if(elemIdx >= n) return;

    if(threadIdx.x == 0)
    {
        _index = index[batchIdx];
    }

    __syncthreads();

    vector[batchIdx * n + elemIdx] = (V)(elemIdx == _index);
}


template<class T>
static __global__ void GenerateDropoutMaskKernel(T *mask, const T *uniformDist,
        const int n, const T dropoutRate)
{
    int idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    mask[idx] = (T)(uniformDist[idx] >= dropoutRate) / ((T)1 - dropoutRate);
}


template<class T>
static __global__ void RmspropKernel(T *newDerivs, const T *derivs, T *msDeriv,
        const T decayRate, const int n)
{
    unsigned long idx;
    T ms, rms, deriv;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    ms = msDeriv[idx];
    deriv = derivs[idx];

    T bound = _sqrt<T>((T)1 / ((T)1 - decayRate));

    ms = decayRate * ms + ((T)1 - decayRate) * deriv * deriv;
    rms = _sqrt<T>(ms) + (T)1e-20;

    newDerivs[idx] = min(bound, max(-bound, deriv / rms));
    msDeriv[idx] = ms;
}


template<class T>
static __global__ void AdadeltaKernel(T *deltas, const T *derivs, T *msDeriv, T *msDelta,
        const T learningRate, const T decayRate, const int n)
{
    unsigned int idx;
    T _msDelta, rmsDelta;
    T _msDeriv, rmsDeriv;
    T deriv, delta;

    const T bound = (T)10;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    _msDeriv = msDeriv[idx];
    _msDelta = msDelta[idx];
    deriv = derivs[idx];

    _msDeriv = decayRate * _msDeriv + ((T)1 - decayRate) * deriv * deriv;
    rmsDeriv = _sqrt<T>(_msDeriv) + (T)1e-20;

    rmsDelta = _sqrt<T>(_msDelta + learningRate * learningRate);

    delta = rmsDelta * min(bound, max(-bound, deriv / rmsDeriv));

    _msDelta = decayRate * _msDelta + ((T)1 - decayRate) * delta * delta;

    deltas[idx] = delta;
    msDeriv[idx] = _msDeriv;
    msDelta[idx] = _msDelta;
}


template<class T>
void MatSet(T *_x, const unsigned long ldx, const T val,
        const unsigned long nRows, const unsigned long nCols, const hipStream_t stream)
{
    dim3 dimGrid((nRows * nCols + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    MatSetKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, ldx, val, nRows, nCols);
}


template<class T>
void MatElemMult(const T *_x, const unsigned long ldx,
        const T *_y, const unsigned long ldy,
        T *_z, const unsigned long ldz,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream)
{
    dim3 dimGrid((nRows * nCols + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    MatElemMultKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, ldx, _y, ldy, _z, ldz, nRows, nCols);
}


template<class T>
void MatAdd(const T *_x, const unsigned long ldx,
        const T *_y, const unsigned long ldy,
        T *_z, const unsigned long ldz,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream)
{
    dim3 dimGrid((nRows * nCols + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    MatAddKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, ldx, _y, ldy, _z, ldz, nRows, nCols);
}


template<class T>
void MatSub(const T *_x, const unsigned long ldx,
        const T *_y, const unsigned long ldy,
        T *_z, const unsigned long ldz,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream)
{
    dim3 dimGrid((nRows * nCols + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    MatSubKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, ldx, _y, ldy, _z, ldz, nRows, nCols);
}


template<class T>
void MatAddToDiag(T *_x, const T val, const unsigned long shift,
        const unsigned long nRows, const hipStream_t stream)
{
    dim3 dimGrid((nRows + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    MatAddToDiagKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, val, shift, nRows);
}


template<class T>
void MatMakeTri(T *_x, const bool upper, const unsigned long nRows, hipStream_t stream)
{
    dim3 dimGrid((nRows * nRows + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    MatMakeTriKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, upper, nRows);
}


template<class T>
void FuncSigmoid(const T *_x, const unsigned long ldx,
        T *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream)
{
    dim3 dimGrid((nRows * nCols + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncSigmoidKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, ldx, _y, ldy, nRows, nCols);
}


template<class T>
void FuncTanh(const T *_x, const unsigned long ldx,
        T *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream)
{
    dim3 dimGrid((nRows * nCols + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncTanhKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, ldx, _y, ldy, nRows, nCols);
}


template<class T>
void FuncSoftplus(const T *_x, const unsigned long ldx,
        T *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream)
{
    dim3 dimGrid((nRows * nCols + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncSoftplusKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, ldx, _y, ldy, nRows, nCols);
}


template<class T>
void FuncRectLinear(const T *_x, const unsigned long ldx,
        T *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream)
{
    dim3 dimGrid((nRows * nCols + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncRectLinearKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, ldx, _y, ldy, nRows, nCols);
}


template<class T>
void FuncSoftmax(const T *_x, const unsigned long ldx,
        T *_y, const unsigned long ldy,
        const unsigned long layerSize, const unsigned long batchSize,
        const hipStream_t stream)
{
    dim3 dimGrid(batchSize);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncSoftmaxKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, ldx, _y, ldy, layerSize);
}


template<class I, class V>
void FuncCTCDecode(const V *_x, const unsigned long ldx,
        V *_y, const unsigned long ldy, const I *_prevIdxMax, I *_idxMax,
        const unsigned long layerSize, const unsigned long nStep,
        const unsigned long nStream, const hipStream_t stream)
{
    dim3 dimGrid(nStream);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncCTCDecodeKernel<I, V><<<dimGrid, dimBlock, 0, stream>>>(_x, ldx, _y, ldy, _prevIdxMax, _idxMax, layerSize, nStep);
}


template<class T>
void FuncBoundRange(const T *_x, const unsigned long ldx,
        T *_y, const unsigned long ldy,
        const T min, const T max,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream)
{
    dim3 dimGrid((nRows * nCols + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncBoundRangeKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, ldx, _y, ldy, min, max, nRows, nCols);
}


template<class T>
void FuncSigmoidDeriv(const T *_x, const unsigned long ldx,
        T *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream)
{
    dim3 dimGrid((nRows * nCols + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncSigmoidDerivKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, ldx, _y, ldy, nRows, nCols);
}


template<class T>
void FuncTanhDeriv(const T *_x, const unsigned long ldx,
        T *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream)
{
    dim3 dimGrid((nRows * nCols + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncTanhDerivKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, ldx, _y, ldy, nRows, nCols);
}


template<class T>
void FuncSoftplusDeriv(const T *_x, const unsigned long ldx,
        T *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream)
{
    dim3 dimGrid((nRows * nCols + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncSoftplusDerivKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, ldx, _y, ldy, nRows, nCols);
}


template<class T>
void FuncRectLinearDeriv(const T *_x, const unsigned long ldx,
        T *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream)
{
    dim3 dimGrid((nRows * nCols + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncRectLinearDerivKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, ldx, _y, ldy, nRows, nCols);
}


template<class I, class V>
void OneHotEncode(const I *_index, V *_vector, const unsigned long n,
        const unsigned long batchSize, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK, batchSize);
    dim3 dimBlock(THREAD_PER_BLOCK);

    OneHotEncodeKernel<I, V><<<dimGrid, dimBlock, 0, stream>>>(_index, _vector, n);
}


template<class T>
void GenerateDropoutMask(T *_mask, const T *_uniformDist, const unsigned long n,
        const T dropoutRate, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    GenerateDropoutMaskKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_mask, _uniformDist, n, dropoutRate);
}


template<class T>
void Rmsprop(T *_newDerivs, const T *_derivs, T *_msDeriv, const T decayRate,
        const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    RmspropKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_newDerivs, _derivs, _msDeriv, decayRate, n);
}


template<class T>
void Adadelta(T *_deltas, const T *_derivs, T *_msDeriv, T *_msDelta,
        const T learningRate, const T decayRate, const unsigned long n,
        const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    AdadeltaKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_deltas, _derivs, _msDeriv, _msDelta, learningRate, decayRate, n);
}


template void MatSet<float>(float *_x, const unsigned long ldx, const float val,
        const unsigned long nRows, const unsigned long nCols, const hipStream_t stream);
template void MatSet<double>(double *_x, const unsigned long ldx, const double val,
        const unsigned long nRows, const unsigned long nCols, const hipStream_t stream);
template void MatSet<int>(int *_x, const unsigned long ldx, const int val,
        const unsigned long nRows, const unsigned long nCols, const hipStream_t stream);
template void MatSet<long>(long *_x, const unsigned long ldx, const long val,
        const unsigned long nRows, const unsigned long nCols, const hipStream_t stream);
template void MatSet<long long>(long long *_x, const unsigned long ldx, const long long val,
        const unsigned long nRows, const unsigned long nCols, const hipStream_t stream);

template void MatElemMult<float>(const float *_x, const unsigned long ldx,
        const float *_y, const unsigned long ldy,
        float *_z, const unsigned long ldz,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);
template void MatElemMult<double>(const double *_x, const unsigned long ldx,
        const double *_y, const unsigned long ldy,
        double *_z, const unsigned long ldz,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);

template void MatAdd<float>(const float *_x, const unsigned long ldx,
        const float *_y, const unsigned long ldy,
        float *_z, const unsigned long ldz,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);
template void MatAdd<double>(const double *_x, const unsigned long ldx,
        const double *_y, const unsigned long ldy,
        double *_z, const unsigned long ldz,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);

template void MatSub<float>(const float *_x, const unsigned long ldx,
        const float *_y, const unsigned long ldy,
        float *_z, const unsigned long ldz,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);
template void MatSub<double>(const double *_x, const unsigned long ldx,
        const double *_y, const unsigned long ldy,
        double *_z, const unsigned long ldz,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);

template void MatAddToDiag<float>(float *_x, const float val, const unsigned long shift,
        const unsigned long nRows, const hipStream_t stream);
template void MatAddToDiag<double>(double *_x, const double val, const unsigned long shift,
        const unsigned long nRows, const hipStream_t stream);

template void MatMakeTri<float>(float *_x, const bool upper,
        const unsigned long nRows, hipStream_t stream);
template void MatMakeTri<double>(double *_x, const bool upper,
        const unsigned long nRows, hipStream_t stream);

template void FuncSigmoid<float>(const float *_x, const unsigned long ldx,
        float *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);
template void FuncSigmoid<double>(const double *_x, const unsigned long ldx,
        double *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);

template void FuncTanh<float>(const float *_x, const unsigned long ldx,
        float *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);
template void FuncTanh<double>(const double *_x, const unsigned long ldx,
        double *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);

template void FuncSoftplus<float>(const float *_x, const unsigned long ldx,
        float *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);
template void FuncSoftplus<double>(const double *_x, const unsigned long ldx,
        double *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);

template void FuncRectLinear<float>(const float *_x, const unsigned long ldx,
        float *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);
template void FuncRectLinear<double>(const double *_x, const unsigned long ldx,
        double *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);

template void FuncSoftmax<float>(const float *_x, const unsigned long ldx,
        float *_y, const unsigned long ldy,
        const unsigned long layerSize, const unsigned long batchSize,
        const hipStream_t stream);
template void FuncSoftmax<double>(const double *_x, const unsigned long ldx,
        double *_y, const unsigned long ldy,
        const unsigned long layerSize, const unsigned long batchSize,
        const hipStream_t stream);

template void FuncCTCDecode<int, float>(const float *_x, const unsigned long ldx,
        float *_y, const unsigned long ldy, const int *_prevIdxMax, int *_idxMax,
        const unsigned long layerSize, const unsigned long nStep,
        const unsigned long nStream, const hipStream_t stream);
template void FuncCTCDecode<long, float>(const float *_x, const unsigned long ldx,
        float *_y, const unsigned long ldy, const long *_prevIdxMax, long *_idxMax,
        const unsigned long layerSize, const unsigned long nStep,
        const unsigned long nStream, const hipStream_t stream);
template void FuncCTCDecode<long long, float>(const float *_x, const unsigned long ldx,
        float *_y, const unsigned long ldy, const long long *_prevIdxMax, long long *_idxMax,
        const unsigned long layerSize, const unsigned long nStep,
        const unsigned long nStream, const hipStream_t stream);
template void FuncCTCDecode<int, double>(const double *_x, const unsigned long ldx,
        double *_y, const unsigned long ldy, const int *_prevIdxMax, int *_idxMax,
        const unsigned long layerSize, const unsigned long nStep,
        const unsigned long nStream, const hipStream_t stream);
template void FuncCTCDecode<long, double>(const double *_x, const unsigned long ldx,
        double *_y, const unsigned long ldy, const long *_prevIdxMax, long *_idxMax,
        const unsigned long layerSize, const unsigned long nStep,
        const unsigned long nStream, const hipStream_t stream);
template void FuncCTCDecode<long long, double>(const double *_x, const unsigned long ldx,
        double *_y, const unsigned long ldy, const long long *_prevIdxMax, long long *_idxMax,
        const unsigned long layerSize, const unsigned long nStep,
        const unsigned long nStream, const hipStream_t stream);

template void FuncBoundRange<float>(const float *_x, const unsigned long ldx,
        float *_y, const unsigned long ldy,
        const float min, const float max,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);
template void FuncBoundRange<double>(const double *_x, const unsigned long ldx,
        double *_y, const unsigned long ldy,
        const double min, const double max,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);

template void FuncSigmoidDeriv<float>(const float *_x, const unsigned long ldx,
        float *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);
template void FuncSigmoidDeriv<double>(const double *_x, const unsigned long ldx,
        double *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);

template void FuncTanhDeriv<float>(const float *_x, const unsigned long ldx,
        float *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);
template void FuncTanhDeriv<double>(const double *_x, const unsigned long ldx,
        double *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);

template void FuncSoftplusDeriv<float>(const float *_x, const unsigned long ldx,
        float *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);
template void FuncSoftplusDeriv<double>(const double *_x, const unsigned long ldx,
        double *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);

template void FuncRectLinearDeriv<float>(const float *_x, const unsigned long ldx,
        float *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);
template void FuncRectLinearDeriv<double>(const double *_x, const unsigned long ldx,
        double *_y, const unsigned long ldy,
        const unsigned long nRows, const unsigned long nCols,
        const hipStream_t stream);

template void OneHotEncode<int, float>(const int *_index, float *_vector, const unsigned long n,
        const unsigned long batchSize, const hipStream_t stream);
template void OneHotEncode<long, float>(const long *_index, float *_vector, const unsigned long n,
        const unsigned long batchSize, const hipStream_t stream);
template void OneHotEncode<long long, float>(const long long *_index, float *_vector, const unsigned long n,
        const unsigned long batchSize, const hipStream_t stream);
template void OneHotEncode<int, double>(const int *_index, double *_vector, const unsigned long n,
        const unsigned long batchSize, const hipStream_t stream);
template void OneHotEncode<long, double>(const long *_index, double *_vector, const unsigned long n,
        const unsigned long batchSize, const hipStream_t stream);
template void OneHotEncode<long long, double>(const long long *_index, double *_vector, const unsigned long n,
        const unsigned long batchSize, const hipStream_t stream);

template void GenerateDropoutMask<float>(float *_mask, const float *_uniformDist, const unsigned long n,
        const float dropoutRate, const hipStream_t stream);
template void GenerateDropoutMask<double>(double *_mask, const double *_uniformDist, const unsigned long n,
        const double dropoutRate, const hipStream_t stream);

template void Rmsprop<float>(float *_newDerivs, const float *_derivs, float *_msDeriv, const float decayRate,
        const unsigned long n, const hipStream_t stream);
template void Rmsprop<double>(double *_newDerivs, const double *_derivs, double *_msDeriv, const double decayRate,
        const unsigned long n, const hipStream_t stream);

template void Adadelta<float>(float *_deltas, const float *_derivs, float *_msDeriv, float *_msDelta,
        const float learningRate, const float decayRate, const unsigned long n,
        const hipStream_t stream);
template void Adadelta<double>(double *_deltas, const double *_derivs, double *_msDeriv, double *_msDelta,
        const double learningRate, const double decayRate, const unsigned long n,
        const hipStream_t stream);

}

}

